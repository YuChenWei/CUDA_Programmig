
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
void sumArrayOnHost(float* A,float* B,float* C,const int nElem)
{
	for(int idx=0;idx<nElem;idx++)
	{
		C[idx]=A[idx]+B[idx];
	}
}
void initialData(float* ip,const int nElem)
{
	time_t t;
	srand((unsigned int)time(&t));
	for(int idx=0;idx<nElem;idx++)
	{
		ip[idx]=(float)(rand()&0xff)/10.0f;
	}
}
void printData(float* A,float* B,float* C,const int nElem)
{
	for(int idx=0;idx<nElem;idx++)
	{
		printf("%f %f %f\n",A[idx],B[idx],C[idx]);
	}
}

int main(int argc,char** argv)
{
	int nElem =1024;
	size_t nBytes=nElem*sizeof(float);
	float* h_A, *h_B, *h_C;
	h_A=(float *)malloc(nBytes);
	h_B=(float *)malloc(nBytes);
	h_C=(float *)malloc(nBytes);
	

	initialData(h_A,nElem);
	initialData(h_B,nElem);
	
	
	sumArrayOnHost(h_A,h_B,h_C,nElem);
	printData(h_A,h_B,h_C,nElem);
	
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
